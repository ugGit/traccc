#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include "matrix_sums.cu"

const size_t MATRIX_SIZE_MIN = 2<<4; // min matrix side dimension
const size_t MATRIX_SIZE_MAX = 2<<13;// max matrix side dimension
const int BLOCK_SIZE_MIN = 1;
const int BLOCK_SIZE_MAX = 2 << 9;  // CUDA maximum is 1024 (i.e. 2<<9)

static void BM_CudaMatrixSum(benchmark::State& state){
  for (auto _ : state){
    cuda::matrix_sums(state.range(0), state.range(1));
  }
}

// Register the function as a benchmark
BENCHMARK(BM_CudaMatrixSum)
  ->ArgsProduct({
    // range for matrix arguments
    {MATRIX_SIZE_MIN, MATRIX_SIZE_MAX},
    // range for thread block size
    {benchmark::CreateRange(BLOCK_SIZE_MIN, BLOCK_SIZE_MAX, /*multi=*/2)}
  });

BENCHMARK_MAIN();
