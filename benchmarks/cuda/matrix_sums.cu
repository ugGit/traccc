
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

namespace cuda{

// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x + blockIdx.x * blockDim.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx*ds + i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}

// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x + blockIdx.x * blockDim.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx + i*ds];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}

bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}

using std::cout;

int matrix_sums(int matrix_size, int block_size){
  // start crono
  // const auto t1 = std::chrono::high_resolution_clock::now();

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[matrix_size*matrix_size];  // allocate space for data in host memory
  h_sums = new float[matrix_size]();
    
  for (int i = 0; i < matrix_size*matrix_size; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
    
  hipMalloc(&d_A, matrix_size*matrix_size*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, matrix_size*sizeof(float)); // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
    
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, matrix_size*matrix_size*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  
  hipMemset(d_sums, 0, matrix_size*sizeof(float));
    
  column_sums<<<(matrix_size+block_size-1)/block_size, block_size>>>(d_A, d_sums, matrix_size);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
    
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, matrix_size*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

  // stop crono
  // const auto t2 = std::chrono::high_resolution_clock::now();
  
  return 0;
}
  
}
